
#include <hip/hip_runtime.h>
#include <stdio.h>

int test_math();
int test_index();
int test_count();

void check(int (*test_func)(), char *func_name, int *ret) {
    printf("%s\n", func_name);
    *ret = test_func();
}

int main() {
    int ret = 0;

    printf("start test\n");
    
    check(test_math, "test_math", &ret);
    check(test_index, "test_index", &ret);
    check(test_count, "test_count", &ret);

    return ret;
}
