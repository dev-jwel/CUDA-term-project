
#include <hip/hip_runtime.h>
#include <stdio.h>

int test_math();
int test_index();
int test_count();
int test_sort();
int test_triangle();

void check(int (*test_func)(), char *func_name, int *ret) {
    printf("%s\n", func_name);
    *ret = test_func();
}

int main() {
    int ret = 0;

    printf("start test\n");
    
    check(test_math, "test_math", &ret);
    check(test_index, "test_index", &ret);
    check(test_count, "test_count", &ret);
    check(test_sort, "test_sort", &ret);
    check(test_triangle, "test_triangle", &ret);

    return ret;
}
