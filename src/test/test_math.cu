#include <stdio.h>
#include "header.cuh"

int main() {
	int i, ret = 0;

	Count_t host_test_1[10] = {1,2,3,4,5,6,7,8,9,10};
	Count_t host_test_2[10] = {2,3,4,5,6,7,8,9,10,11};
	Count_t host_acc_sum[10] = {1,3,6,10,15,21,28,36,45,55};
	Count_t host_mul[10] = {2,6,12,20,30,42,56,72,90,110};
	Count_t host_result[10];

	Count_t *dev_test_1, *dev_test_2, *dev_result;

	hipMalloc((void **) &dev_test_1, sizeof(host_test_1));
	hipMalloc((void **) &dev_test_2, sizeof(host_test_2));
	hipMalloc((void **) &dev_result, sizeof(host_result));

	hipMemcpy(dev_test_1, host_test_1, sizeof(host_test_1), cudaHostToDevice);
	hipMemcpy(dev_test_2, host_test_2, sizeof(host_test_2), cudaHostToDevice);
	
	acc_sum <<<1, BLOCK_SIZE>>> (dev_test_1, dev_result, 10);
	hipMemcpy(host_result, dev_result, sizeof(host_result), cudaDeviceToHost);
	for (i=0; i<10; ++i) {
		if (host_result[i] != host_acc_sum[i]) {
			printf("error on accsum\n");
			ret = 1;
			break;
		}
	}

	mul <<<1, BLOCK_SIZE>>> (dev_test_1, dev_test_2, dev_result, 10);
	hipMemcpy(host_result, dev_result, sizeof(host_result), cudaDeviceToHost);
	for (i=0; i<10; ++i) {
		if (host_result[i] != host_mul[i]) {
			printf("error on mul\n");
			ret = 1;
			break;
		}
	}

	sum <<<1, BLOCK_SIZE>>> (dev_test_1, dev_result, 10);
	hipMemcpy(host_result, dev_result, sizeof(host_result), cudaDeviceToHost);
	if (dev_result[0] != 55) {
		printf("error on accsum\n");
		ret = 1;
	}

	hipFree(dev_test_1);
	hipFree(dev_test_2);
	hipFree(dev_result);

	return ret;
}