#include <fstream>
#include <sstream>
#include <string>
#include <vector>
#include "header.cuh"

using namespace std;

vector<Edge> load_bitcoin_otc() {
	ifstream ifs("soc-sign-bitcoinotc.csv");
	string line;
	string delim = ",";
	vector<Edge> edge_list;

	while (get_line(ifs, line)) {
		string src = line.substr(0, line.find(delim);
		line.erase(0, line.find(delim)+1);
		string dst = line.substr(0, line.find(delim);
		edge_list.push_back({stoi(src), stoi(dst)});
	}

	return edge_list;
}