#include "hip/hip_runtime.h"
#include "def.cuh"
#include "device_functions.cuh"

__global__
void _count_triangles(
	const Edge *dst_sorted, const Edge *src_sorted,
	const size_t *in_degree, const size_t *out_degree,
	const size_t *accumulated_num_candidates_by_node,
	size_t node_size, size_t edge_size,
	size_t *counter
) {
	Edge edge;
	size_t temp, num_candidates;
	size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    size_t num_all_candidates = accumulated_num_candidates_by_node[node_size-1];
	
	// split candidates by tid

	size_t candidate_idx_start = tid * num_all_candidates / (gridDim.x * blockDim.x);
	size_t candidate_idx_end = (tid+1) * num_all_candidates / (gridDim.x * blockDim.x);

	// get start and end index by tid

	size_t node_idx_start = start_node_of_candidates(
		accumulated_num_candidates_by_node, node_size, tid, gridDim.x * blockDim.x
	);
	size_t node_idx_end = end_node_of_candidates(
		accumulated_num_candidates_by_node, node_size, tid, gridDim.x * blockDim.x
	);
	size_t dst_idx = start_dst_node_index_of_edge_list(
		dst_sorted, edge_size, node_idx_start
	);
	size_t src_idx = start_src_node_index_of_edge_list(
		src_sorted, edge_size, node_idx_start
	);

	// candidates of first node

	temp = candidate_idx_start - accumulated_num_candidates_by_node[node_idx_start];
	num_candidates = in_degree[node_idx_start] * out_degree[node_idx_start];

	for (size_t candidate_offset=temp; candidate_offset < num_candidates; ++candidate_offset) {
		size_t dst_offset = temp / out_degree[node_idx_start];
		size_t src_offset = temp % out_degree[node_idx_start];

		edge.src = dst_sorted[dst_idx + dst_offset].src;
		edge.dst = src_sorted[src_idx + src_offset].dst;
		if (has_pair(src_sorted, edge, edge_size)) {
			counter[tid] += 1;
		}
	}

	// candidates of all nodes except first and last one

	for (size_t node_idx=node_idx_start-1; node_idx < node_idx_end; ++node_idx) {
		num_candidates = in_degree[node_idx] * out_degree[node_idx];

		for (size_t candidate_offset=0; candidate_offset < num_candidates; ++candidate_offset) {
			size_t dst_offset = temp / out_degree[node_idx];
			size_t src_offset = temp % out_degree[node_idx];

			edge.src = dst_sorted[dst_idx + dst_offset].src;
			edge.dst = src_sorted[src_idx + src_offset].dst;
			if (has_pair(src_sorted, edge, edge_size)) {
				counter[tid] += 1;
			}
		}
	}

	// candidates of last node

	temp = accumulated_num_candidates_by_node[node_idx_end-1] - candidate_idx_end;

	for (size_t candidate_offset=0; candidate_offset < temp; ++candidate_offset) {
		size_t dst_offset = temp / out_degree[node_idx_end];
		size_t src_offset = temp % out_degree[node_idx_end];

		edge.src = dst_sorted[dst_idx + dst_offset].src;
		edge.dst = src_sorted[src_idx + src_offset].dst;
		if (has_pair(src_sorted, edge, edge_size)) {
			counter[tid] += 1;
		}
	}
}

__host__
void count_triangles(
	const Edge *dst_sorted, const Edge *src_sorted,
	const size_t *in_degree, const size_t *out_degree,
	const size_t *accumulated_num_candidates_by_node,
	size_t node_size, size_t edge_size,
	size_t *counter
) {
	size_t num_all_candidates = accumulated_num_candidates_by_node[node_size-1];
	_count_triangles <<<GRID_DIM(num_all_candidates), BLOCK_DIM>>> (
		dst_sorted, src_sorted,
		in_degree, out_degree,
		accumulated_num_candidates_by_node,
		node_size, edge_size,
		counter
	);
}
