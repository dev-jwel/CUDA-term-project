#include "hip/hip_runtime.h"
#include "def.cuh"
#include <stdio.h>

__device__
size_t binary_search(
	const void *array,
	int (*compare) (const void *, const void *),
	size_t element_size,
	size_t array_size,
	const void *target,
	bool select_left
) {
	size_t min = 0;
	size_t max = array_size;
	size_t mid;

	while (min < max) {
		mid = (min + max) / 2;
		switch (compare(array + element_size * mid, target)) {
			case -1: min = mid + 1; break;
			case  1: max = mid; break;
			case  0:
				if (select_left) {
					max = mid;
				} else {
					min = mid + 1;
				}
			break;
		}
	}

	return min;
}

__device__
int compare_src(const void *edge1, const void *edge2) {
	if (((Edge *)edge1)->src < ((Edge *)edge2)->src) return -1;
	if (((Edge *)edge1)->src > ((Edge *)edge2)->src) return 1;
	return 0;
}

__device__
int compare_dst(const void *edge1, const void *edge2) {
	if (((Edge *)edge1)->dst < ((Edge *)edge2)->dst) return -1;
	if (((Edge *)edge1)->dst > ((Edge *)edge2)->dst) return 1;
	return 0;
}
__device__
int compare_edge(const void *edge1, const void *edge2) {
	if (((Edge *)edge1)->src < ((Edge *)edge2)->src) return -1;
	if (((Edge *)edge1)->src > ((Edge *)edge2)->src) return 1;
	if (((Edge *)edge1)->dst < ((Edge *)edge2)->dst) return -1;
	if (((Edge *)edge1)->dst > ((Edge *)edge2)->dst) return 1;
	return 0;
}

__device__
int compare_count(const void *cnt1, const void *cnt2) {
	if (*((size_t *) cnt1) < *((size_t *) cnt2)) {
		return -1;
	} else if (*((size_t *) cnt1) > *((size_t *) cnt2)) {
		return 1;
	} else {
		return 0;
	}
}

__device__
size_t start_src_node_index_of_edge_list(
	const Edge *edges,
	size_t edge_size,
	size_t idx
) {
	Edge target = {idx, 0};
	return binary_search(edges, compare_src, sizeof(Edge), edge_size, (void *) &target, true);
}


__device__
size_t start_dst_node_index_of_edge_list(
	const Edge *edges,
	size_t edge_size,
	size_t idx
) {
	Edge target = {0, idx};
	return binary_search(edges, compare_dst, sizeof(Edge), edge_size, (void *) &target, true);
}

__device__
size_t start_node_of_candidates(
	const size_t *accumulated_num_candidates_by_node,
	size_t node_size,
	size_t tid,
	size_t num_threads
) {
	size_t num_all_candidates = accumulated_num_candidates_by_node[node_size-1];
	size_t target = tid * num_all_candidates / num_threads + 1;

	size_t ret = binary_search(
		accumulated_num_candidates_by_node, compare_count, sizeof(size_t), node_size, (void *) &target, true
	);

	if (accumulated_num_candidates_by_node[ret] != target) {
		ret += 1;
	}

	return ret;
}

__device__
size_t end_node_of_candidates(
	const size_t *accumulated_num_candidates_by_node,
	size_t node_size,
	size_t tid,
	size_t num_threads
) {
	size_t num_all_candidates = accumulated_num_candidates_by_node[node_size-1];
	size_t target = (tid+1) * num_all_candidates / num_threads;

	size_t ret = binary_search(
		accumulated_num_candidates_by_node, compare_count, sizeof(size_t), node_size, (void *) &target, true
	);

	if (accumulated_num_candidates_by_node[ret] != target) {
		ret += 1;
	}

	return ret;
}

__device__
bool has_pair(const Edge *fully_sorted_edge, Edge edge, size_t edge_size) {
	Edge target = edge;
	size_t idx = binary_search(fully_sorted_edge, compare_edge, sizeof(Edge), edge_size, (void *) &target, true);
	return compare_edge((void *) &fully_sorted_edge[idx], (void *) &target) == 0;
}
