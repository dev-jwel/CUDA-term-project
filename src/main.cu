#include <stdio.h>
#include "header.cuh"

int main(int argc, char *argv[]) {
	printf("hello\n");

	struct Edge *dev_c;
    struct Edge dev;

    hipMalloc((void**)&dev_c, sizeof(Edge)*N);
    sort_by_dest<<< >>>(dev_c, N); // <<< >>> 안에 뭐 넣어야 하는지 모르겠음
    hipMemcpy(&dev, dev_c, sizeof(Edge)*N, hipMemcpyDeviceToHost); // dst에 대해 정렬

    hipMalloc((void**)&dev_c, sizeof(Edge)*N);
    stable_sort_by_source<<< >>>(dev_c, N); // <<< >>> 안에 뭐 넣어야 하는지 모르겠음
    hipMemcpy(&dev, dev_c, sizeof(Edge)*N, hipMemcpyDeviceToHost); // src에 대해 정렬
	return 0;
}
