#include <iostream>
#include <vector>
#include "def.cuh"
#include "host_functions.cuh"

#define edge_size 512 // Edge 개수

using namespace std;


int main(int argc, char *argv[]) {
	srand(time(NULL));

	size_t max_node_idx = 0;
	Edge *dev_dst, *dev_dst_final, *buffer;
	Edge *dev_src, *dev_src_final;
	Edge *host_src_result;
	vector<Edge> edge_list = load_bitcoin_otc();

	for (auto &edge : edge_list) {
		if (edge.src > max_node_idx) {
			max_node_idx = edge.src;
		}
		if (edge.dst > max_node_idx) {
			max_node_idx = edge.dst;
		}
	}

	size_t host_result_in_degree[max_node_idx], host_result_out_degree[max_node_idx];
	size_t *dev_result_degree;

	hipMalloc((void **)&dev_dst, sizeof(edge_list));
	hipMalloc((void **)&dev_src, sizeof(edge_list));
	hipMalloc((void **)&dev_dst_final, sizeof(edge_list));
	hipMalloc((void **)&buffer, sizeof(edge_list));
	hipMalloc((void **)&dev_src_final, sizeof(edge_list));

	hipMemcpy(dev_dst, edge_list, sizeof(edge_list), hipMemcpyHostToDevice);
	sort_by_dst(dev_dst, dev_dst_final, buffer, edge_list.size());

	hipMemcpy(dev_src, dev_dst_final, sizeof(edge_list), hipMemcpyDeviceToDevice);
	stable_sort_by_src(dev_src, dev_src_final, buffer, edge_list.size());

	host_src_result = (edge_list*)malloc(sizeof(edge_list));
	hipMemcpy(host_src_result, dev_result, sizeof(edge_list), hipMemcpyDeviceToHost);
	// 위에까지 sort 함수

	hipMalloc((void **) &dev_result_degree, sizeof(host_result_in_degree));

	count_in_degree(dev_dst_final, dev_result_degree, edge_list, max_node_idx);
	hipMemcpy(host_result_in_degree, dev_result_degree, sizeof(host_result_in_degree), hipMemcpyDeviceToHost);
	
	count_out_degree(dev_src_final, dev_result_degree, edge_list, max_node_idx);
	hipMemcpy(host_result_out_degree, dev_result_degree, sizeof(host_result_out_degree), hipMemcpyDeviceToHost);
	//위에까지 count_degree 함수

	

	



	cout << "max_node_idx: " << max_node_idx << ", num_edge: " << edge_list.size() << endl;
    cout << "count: " << naive_counter(edge_list) << endl;



	return 0;
}
