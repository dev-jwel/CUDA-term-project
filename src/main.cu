#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include "header.cuh"

#define edge_size 100 // Edge 개수

int main(int argc, char *argv[]) {
	srand(time(NULL));
	printf("hello\n");
	int block_size, edge_size, grid_size, node_size;
	struct Edge *dev_in;
	struct Edge *dev_out;
	struct Edge *dev_out2;
	
	block_size = BLOCK_SIZE;
	grid_size = ceil(edge_size / block_size);

	struct Edge *in = (struct Edge *)malloc(sizeof(struct Edge)*edge_size);
	struct Edge *out = (struct Edge *)malloc(sizeof(struct Edge)*edge_size);
	struct Edge *out2 = (struct Edge *)malloc(sizeof(struct Edge)*edge_size);
	Count_t *c_in_out = (Count_t *)malloc(node_size * sizeof(Count_t));
	Count_t *c_out_out = (Count_t *)malloc(node_size * sizeof(Count_t)); 

	hipMalloc((void**)&dev_in, sizeof(Edge)*edge_size);
	hipMalloc((void**)&dev_out, sizeof(Edge)*edge_size);
	hipMalloc((void**)&dev_out2, sizeof(Edge)*edge_size);
	hipMalloc((void**)&dev_c_in_out, sizeof(Count_t)*node_size);
	hipMalloc((void**)&dev_c_out_out, sizeof(Count_t)*node_size);

	dim3 Dg(grid_size, 1, 1);
	dim3 Db(block_size, 1, 1)
	for(int i=0; i<edge_size; i++) {
		sort_by_dest<<<Dg, Db>>>(dev_in, dev_out, edge_size);
	}
	hipMemcpy(&out, dev_out, sizeof(Edge)*edge_size, hipMemcpyDeviceToHost); // dst에 대해 정렬된 값 src로

	for(int j=0; j<edge_size; j++) {
		stable_sort_by_source<<<Dg, Db>>>(dev_out, dev_out2, edge_size);
	}

	hipMemcpy(&out2, dev_out2, sizeof(Edge)*edge_size, hipMemcpyDeviceToHost); // src에 대해 정렬된 값 host로

	initial_out(&cout, node_size); // cout 초기화 과정

	count_in_degree<<Dg, Db>>>(dev_out, dev_c_in_out, edge_size, node_size);
	count_out_degree<<<Dg, Db>>>(dev_out2, dev_c_out_out, edge_size, node_size);

	hipMemcpy(&c_in_out, dev_c_in_out, sizeof(Count_t)*node_size, hipMemcpyDeviceToHost); // dst 기준으로 정렬된 outgoing 차수 device에서 host로
	hipMemcpy(&c_out_out, dev_c_out_out, sizeof(Count_t)*node_size, hipMemcpyDeviceToHost); // src 기준으로 정렬된 incoming 차수 device에서 host로
	//hipFree(dev_out);




	hipDeviceSynchronize(); //device가 작업 완료할 때까지 host는 대기.
	return 0;
}
